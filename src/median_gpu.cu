#include "hip/hip_runtime.h"
#include "median_gpu.hpp"
#include <stdio.h>
#include <thrust/sort.h>

const int BLOCKDIM = 16;
//__device__ const int FILTER_SIZE = 9;
//__device__ const int FILTER_HALFSIZE = FILTER_SIZE >> 1;

__device__ void sort_quick(uint8_t *x, int left_idx, int right_idx)
{
	int i = left_idx, j = right_idx;
	uint8_t pivot = x[(left_idx + right_idx) / 2];
	while (i <= j)
	{
		while (x[i] < pivot)
			i++;
		while (x[j] > pivot)
			j--;
		if (i <= j)
		{
			uint8_t temp;
			temp = x[i];
			x[i] = x[j];
			x[j] = temp;
			i++;
			j--;
		}
	};
	if (left_idx < j)
		sort_quick(x, left_idx, j);
	if (i < right_idx)
		sort_quick(x, i, right_idx);
}

__device__ void sort_bubble(uint8_t *x, int n_size)
{
	for (int i = 0; i < n_size - 1; i++)
	{
		for (int j = 0; j < n_size - i - 1; j++)
		{
			if (x[j] > x[j + 1])
			{
				uint8_t temp = x[j];
				x[j] = x[j + 1];
				x[j + 1] = temp;
			}
		}
	}
}

__device__ void sort_insertion(uint8_t *x, int n_size)
{
	for (int k = 1; k < n_size; k++)
	{
		int temp = x[k];
		int j = k - 1;
		while (j >= 0 && temp <= x[j])
		{
			x[j + 1] = x[j];
			j = j - 1;
		}
		x[j + 1] = temp;
	}
}

__device__ void sort_linear(float *x, int n_size)
{
	for (int i = 0; i < n_size - 1; i++)
	{
		int min_idx = i;
		for (int j = i + 1; j < n_size; j++)
		{
			if (x[j] < x[min_idx])
				min_idx = j;
		}
		float temp = x[min_idx];
		x[min_idx] = x[i];
		x[i] = temp;
	}
}

__device__ void swap(uint8_t &a, uint8_t &b)
{
   uint8_t temp; 
   temp = a; 
   a = b; 
   b = temp; 
}

const int ipt = 8;
const int tpb = 128;
const int blks = 1;


__global__ void sort_kernel(uint8_t *windowMedian)
{
	// Specialize BlockRadixSort for a 1D block of 128 threads owning 8 integer items each
	typedef hipcub::BlockRadixSort<uint8_t, tpb, ipt> BlockRadixSort;
	// Allocate shared memory for BlockRadixSort
	__shared__ typename BlockRadixSort::TempStorage temp_storage;
	// Obtain a segment of consecutive items that are blocked across threads
	uint8_t thread_keys[ipt];

	for (int k = 0; k < ipt; k++)
	{
		// printf("\n %d", windowMedian[threadIdx.x * ipt + k]);
		thread_keys[k] = windowMedian[threadIdx.x * ipt + k];
	}
	// Collectively sort the keys
	BlockRadixSort(temp_storage).Sort(thread_keys);
	__syncthreads();
	// write results to output array
	for (int k = 0; k < ipt; k++)
		windowMedian[threadIdx.x * ipt + k] = thread_keys[k];
}


__global__ void temporal_median_filter(uint8_t **recordDEV,
									   uint8_t *src_ptr, int src_pitch,
									   uint8_t *dst_ptr, int dst_pitch,
									   int dst_width, int dst_height,
									   int color_component)
{
	//printf("kernel >>>>>\n");

	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	// printf("kernel >>>> x: %d y: %d dst_width: %d dst_height: %d  color_component: %d\n ", x,y, dst_width, dst_height, color_component);

	// uint8_t windowMedian[RECORD_LENGTH];
	uint8_t windowMedian[RECORD_LENGTH];

	if ((x < dst_width) && (y < dst_height))
	{
		int dst_offset = ((dst_pitch * y) + x * 4) + color_component;
		int src_offset = ((src_pitch * y) + x * 4) + color_component;

		int windowElements;

		for (windowElements = 0; windowElements < RECORD_LENGTH; windowElements++)
		{
			windowMedian[windowElements] = *(recordDEV[windowElements] + dst_offset);
			// printf(" %d, %d \n", windowMedian[windowElements] , *(recordDEV[windowElements] + dst_offset));
		}

		//sort_insertion(windowMedian,windowElements);
		
		// for 128 frame -> Time taken: 0.32s
		//thrust::sort(thrust::device, windowMedian, windowMedian + windowElements);
        
		// for 128 frame -> Time taken: 0.04s
		sort_bubble(windowMedian, windowElements);
		
		// sort_linear(windowMedian,windowElements);
		// sort_quick(windowMedian,0,windowElements);

		//sort_kernel<<<blks,tpb>>>(windowMedian);

		// printf("%d \n", (int)windowMedian[windowElements/2]);
		*(dst_ptr + dst_offset) = windowMedian[windowElements/2];
	}
}

extern "C" void median_filter(NvBufSurface *src, NvBufSurface *dst,
							  std::vector<NvBufSurface *> record)
{

	uint8_t *recordCPU[RECORD_LENGTH];

	for (int i = 0; i < RECORD_LENGTH; i++)
	{
		recordCPU[i] = (uint8_t *)record[i]->surfaceList->dataPtr;
	}

	uint8_t **recordDEV;
	hipMalloc((uint8_t **)&recordDEV, RECORD_LENGTH * sizeof(uint8_t *));
	hipMemcpy(recordDEV, recordCPU, RECORD_LENGTH * sizeof(uint8_t *), hipMemcpyHostToDevice);

	uint8_t *src_ptr = (uint8_t *)src->surfaceList[0].dataPtr;
	int src_pitch = src->surfaceList[0].pitch;

	uint8_t *dst_ptr = (uint8_t *)dst->surfaceList[0].dataPtr;
	int output_cols = dst->surfaceList[0].width;
	int output_rows = dst->surfaceList[0].height;
	int dst_pitch = dst->surfaceList[0].pitch;

	printf("output_rows: %d output_cols: %d dst_pitch: %d src_pitch: %d \n ", output_rows, output_cols, dst_pitch, src_pitch);

	const dim3 block(BLOCKDIM, BLOCKDIM);
	const dim3 grid(output_cols / BLOCKDIM, output_rows / BLOCKDIM);

	for (int color_componenet = 0; color_componenet <= 4; color_componenet++)
	{
		temporal_median_filter<<<grid, block>>>(recordDEV,
												src_ptr, src_pitch,
												dst_ptr, dst_pitch,
												output_cols, output_rows,
												color_componenet);
	}

	hipDeviceSynchronize();

	hipFree(recordDEV);
}
