#include "hip/hip_runtime.h"
#include "median_gpu.hpp"
#include <stdio.h>
#include <thrust/sort.h>


const int BLOCKDIM = 8;
//__device__ const int FILTER_SIZE = 9;
//__device__ const int FILTER_HALFSIZE = FILTER_SIZE >> 1;

__device__ void sort_quick(uint8_t *x, int left_idx, int right_idx)
{
	int i = left_idx, j = right_idx;
	uint8_t pivot = x[(left_idx + right_idx) / 2];
	while (i <= j)
	{
		while (x[i] < pivot)
			i++;
		while (x[j] > pivot)
			j--;
		if (i <= j)
		{
			uint8_t temp;
			temp = x[i];
			x[i] = x[j];
			x[j] = temp;
			i++;
			j--;
		}
	};
	if (left_idx < j)
		sort_quick(x, left_idx, j);
	if (i < right_idx)
		sort_quick(x, i, right_idx);
}

__device__ void sort_bubble(uint8_t *x, int n_size)
{
	for (int i = 0; i < n_size - 1; i++)
	{
		for (int j = 0; j < n_size - i - 1; j++)
		{
			if (x[j] > x[j + 1])
			{
				uint8_t temp = x[j];
				x[j] = x[j + 1];
				x[j + 1] = temp;
			}
		}
	}
}


__device__ void sort_insertion(uint8_t* x , int n_size)
{
    for(int k=1; k<n_size; k++)   
    {  
        int temp = x[k];  
        int j= k-1;  
        while(j>=0 && temp <= x[j])  
        {  
            x[j+1] = x[j];   
            j = j-1;  
        }  
        x[j+1] = temp;  
    } 

}
__device__ void sort_linear(float *x, int n_size)
{
	for (int i = 0; i < n_size - 1; i++)
	{
		int min_idx = i;
		for (int j = i + 1; j < n_size; j++)
		{
			if (x[j] < x[min_idx])
				min_idx = j;
		}
		float temp = x[min_idx];
		x[min_idx] = x[i];
		x[i] = temp;
	}
}

//#include <hipcub/hipcub.hpp>
#include <stdio.h>

const int ipt=8;
const int tpb=128;
const int blks = 1; 


__global__ void sort_kernel(uint8_t* windowMedian)
{
    // Specialize BlockRadixSort for a 1D block of 128 threads owning 8 integer items each
    typedef hipcub::BlockRadixSort<uint8_t, tpb, ipt> BlockRadixSort;
    // Allocate shared memory for BlockRadixSort
    __shared__ typename BlockRadixSort::TempStorage temp_storage;
    //Obtain a segment of consecutive items that are blocked across threads
    uint8_t thread_keys[ipt];

    for (int k = 0; k < ipt; k++) 
    {   
		//printf("\n %d", windowMedian[threadIdx.x * ipt + k]); 
        thread_keys[k] = windowMedian[threadIdx.x * ipt + k];
    }
    // Collectively sort the keys
    BlockRadixSort(temp_storage).Sort(thread_keys);
    __syncthreads();
    // write results to output array
    for (int k = 0; k < ipt; k++) 
       windowMedian[threadIdx.x * ipt + k] = thread_keys[k];
}



__global__ void across_frame_median_filter(uint8_t **recordDEV, int RECORD_LENGTH,
										   uint8_t *src_ptr, int src_pitch,
										   uint8_t *dst_ptr, int dst_pitch,
										   int dst_width, int dst_height,
										   int color_component)
{
    //printf("kernel >>>>>\n");

	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
    //printf("kernel >>>> x: %d y: %d dst_width: %d dst_height: %d  color_component: %d\n ", x,y, dst_width, dst_height, color_component);
    
	uint8_t *windowMedian = (uint8_t*)malloc(RECORD_LENGTH * sizeof(uint8_t)); 

	if ((x < dst_width) && (y < dst_height))
	{
		int dst_offset = ((dst_pitch * y) + x * 4) + color_component;
		int src_offset = ((src_pitch * y) + x * 4) + color_component;

		int windowElements;

		for (windowElements = 0; windowElements < RECORD_LENGTH; windowElements++)
		{
			windowMedian[windowElements] = *(recordDEV[windowElements] + dst_offset);
		}

		
		sort_insertion(windowMedian,windowElements);

		for (windowElements = 0; windowElements < RECORD_LENGTH; windowElements++)
		{
			*(recordDEV[windowElements] + dst_offset) = windowMedian[windowElements]; 
		}

		free(windowMedian); 
	}
}


extern "C" void median_filter(NvBufSurface *src, NvBufSurface *dst,
							  std::vector<NvBufSurface *> record)
{
    int RECORD_LENGTH = record.size();

	uint8_t **recordCPU = (uint8_t **) malloc( RECORD_LENGTH* sizeof(uint8_t *));

	for (int i = 0; i < RECORD_LENGTH; i++)
	{
		recordCPU[i] = (uint8_t *)record[i]->surfaceList->dataPtr;
	}

	uint8_t **recordDEV;
	hipMalloc((uint8_t **)&recordDEV, RECORD_LENGTH * sizeof(uint8_t *));
	hipMemcpy(recordDEV, recordCPU, RECORD_LENGTH * sizeof(uint8_t *), hipMemcpyHostToDevice);

	uint8_t *src_ptr = (uint8_t *)src->surfaceList[0].dataPtr;
	int src_pitch = src->surfaceList[0].pitch;

	uint8_t *dst_ptr = (uint8_t *)dst->surfaceList[0].dataPtr;
	int output_cols = dst->surfaceList[0].width;
	int output_rows = dst->surfaceList[0].height;
	int dst_pitch = dst->surfaceList[0].pitch;

	printf("output_rows: %d output_cols: %d dst_pitch: %d src_pitch: %d \n ", output_rows, output_cols, dst_pitch, src_pitch);

	const dim3 block(BLOCKDIM, BLOCKDIM);
	const dim3 grid(output_cols / BLOCKDIM, output_rows / BLOCKDIM);



	for (int color_componenet = 0; color_componenet <= 4; color_componenet++)
	{
		across_frame_median_filter<<<grid, block>>>(recordDEV, RECORD_LENGTH,  
												  src_ptr, src_pitch,
												  dst_ptr, dst_pitch,
												  output_cols, output_rows,
												  color_componenet);
	}
	
	hipDeviceSynchronize();
    free(recordCPU);
	hipFree(recordDEV);
}




// from here: https://stackoverflow.com/questions/64441827/cuda-thrustsort-met-memory-problem-when-i-still-have-enough-memory
// hipError_t err = hipDeviceSetLimit(hipLimitMallocHeapSize, 1048576ULL*1024);

// __global__ void across_frame_median_filter(uint8_t **recordDEV, uint8_t *windowMedianDEV,
// 										   uint8_t *src_ptr, int src_pitch,
// 										   uint8_t *dst_ptr, int dst_pitch,
// 										   int dst_width, int dst_height,
// 										   int color_component)
// {
//     //printf("kernel >>>>>\n");

// 	const int x = blockIdx.x * blockDim.x + threadIdx.x;
// 	const int y = blockIdx.y * blockDim.y + threadIdx.y;
//     //printf("kernel >>>> x: %d y: %d dst_width: %d dst_height: %d  color_component: %d\n ", x,y, dst_width, dst_height, color_component);

// 	if ((x < dst_width) && (y < dst_height))
// 	{
// 		int dst_offset = ((dst_pitch * y) + x * 4) + color_component;
// 		int src_offset = ((src_pitch * y) + x * 4) + color_component;

// 		float windowMedian[RECORD_LENGTH];
// 		int windowElements;
//         // calling a __host__ function("thrust::device_vector<unsigned char,  ::thrust::device_allocator<unsigned char> > ::device_vector(unsigned long)") from a __global__ function("across_frame_median_filter") is not allowed
//         //thrust::device_vector<uint8_t> d_windowMedian(RECORD_LENGTH);

// 		for (windowElements = 0; windowElements < RECORD_LENGTH; windowElements++)
// 		{
// 			windowMedian[windowElements] = *(recordDEV[windowElements] + dst_offset);
// 		    //d_windowMedian.push_back(*(recordDEV[windowElements] + dst_offset));
// 	        //windowMedianDEV[windowElements] = *(recordDEV[windowElements] + dst_offset);

// 		}

// 	  	// hipError_t status;
// 		// void* tmpStorage = 0;
// 		// size_t tmpStorageSize = 0;
// 		// uint8_t* d_keys = 0;
// 	    // unsigned dataSize = windowElements * sizeof(uint8_t);

// 		//allocateDeviceMemory( &d_keys , dataSize , __LINE__ );
// 	    //copyDataToDevice( d_keys , windowMedian , dataSize , __LINE__ );

// 		//status = hipcub::DeviceRadixSort::SortKeys(tmpStorage, tmpStorageSize, d_keys, d_keys, windowElements);
// 		//CHECK_ERROR( status );

// 		//allocateDeviceMemory( &tmpStorage , tmpStorageSize , __LINE__ );

// 		//status = hipcub::DeviceRadixSort::SortKeys(tmpStorage, tmpStorageSize, d_keys, d_keys, windowElements);
// 		//CHECK_ERROR( status );

// 		//copyDataToHost( h_keys , d_keys , dataSize , __LINE__ );

// 		thrust::sort(thrust::device, windowMedian, windowMedian + windowElements);
// 		//thrust::sort(thrust::device, windowMedianDEV, windowMedianDEV + windowElements);
// 		//thrust::sort(d_windowMedian.begin(), d_windowMedian.end());

// 		//sort_bubble(windowMedian, windowElements);
// 		//sort_linear(windowMedian,windowElements);
// 		///sort_quick(windowMedian,0,windowElements);
// 		//*(dst_ptr + dst_offset) = windowMedianDEV[windowElements / 2];
// 		//*(dst_ptr + dst_offset) = 0; //windowMedianGPU[0];
// 	    *(dst_ptr + dst_offset) = windowMedian[windowElements / 2];

// 	}
// }