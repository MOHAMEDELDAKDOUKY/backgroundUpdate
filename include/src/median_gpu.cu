#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

const int BLOCKDIM = 32;
const int MAX_WINDOW = 11;
//__device__ const int FILTER_SIZE = 9;
//__device__ const int FILTER_HALFSIZE = FILTER_SIZE >> 1;

__device__ void sort_quick(float *x, int left_idx, int right_idx) 
{
      int i = left_idx, j = right_idx;
      float pivot = x[(left_idx + right_idx) / 2];
      while (i <= j) 
      {
            while (x[i] < pivot)
                  i++;
            while (x[j] > pivot)
                  j--;
            if (i <= j) {
		  float temp;
                  temp = x[i];
                  x[i] = x[j];
                  x[j] = temp;
                  i++;
                  j--;
            }
      };
      if (left_idx < j)
            sort_quick(x, left_idx, j);
      if (i < right_idx)
            sort_quick(x, i, right_idx);
}

__device__ void sort_bubble(float *x, int n_size) 
{
	for (int i = 0; i < n_size - 1; i++) 
	{
		for(int j = 0; j < n_size - i - 1; j++) 
		{
			if (x[j] > x[j+1]) 
			{
				float temp = x[j];
				x[j] = x[j+1];
				x[j+1] = temp;
			}
		}
	}
}

__device__ void sort_linear(float *x, int n_size) 
{
	for (int i = 0; i < n_size-1; i++) 
	{
		int min_idx = i;
		for (int j = i + 1; j < n_size; j++) 
		{
			if(x[j] < x[min_idx])
				min_idx = j;
		}
		float temp = x[min_idx];
		x[min_idx] = x[i];
		x[i] = temp;
	}
}


__device__ int index(int x, int y, int width) 
{
	return (y * width) + x;
}


__global__ void median_filter_2d(unsigned char* input, unsigned char* output, int width, int height)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
    printf("cuda+++++++++++++++++++++++++++++") ; 
	if((x<width) && (y<height))
	{
		const int color_tid = index(x,y,width);
		float windowMedian[MAX_WINDOW*MAX_WINDOW];
		int windowElements = 0;

			windowMedian[windowElements] = input[index(x,y,width)];
			windowMedian[windowElements++] = input[index(x,y,width)];
       
		sort_bubble(windowMedian,windowElements);
		//sort_linear(windowMedian,windowElements);
		//sort_quick(windowMedian,0,windowElements);
		output[color_tid] = windowMedian[windowElements/2];
	}
}

void median_filter_wrapper(const cv::Mat& input, cv::Mat& output)
{
	unsigned char *d_input, *d_output;
	
	hipError_t cudaStatus;	
	
	cudaStatus = hipMalloc<unsigned char>(&d_input,input.rows*input.cols);
	checkCudaErrors(cudaStatus);	
	cudaStatus = hipMalloc<unsigned char>(&d_output,output.rows*output.cols);
	checkCudaErrors(cudaStatus);

	cudaStatus = hipMemcpy(d_input,input.ptr(),input.rows*input.cols,hipMemcpyHostToDevice);
	checkCudaErrors(cudaStatus);	
	
	const dim3 block(BLOCKDIM,BLOCKDIM);
	const dim3 grid(input.cols/BLOCKDIM, input.rows/BLOCKDIM);

	median_filter_2d<<<grid,block>>>(d_input,d_output,input.cols,input.rows);

	cudaStatus = hipDeviceSynchronize();
	checkCudaErrors(cudaStatus);	

	cudaStatus = hipMemcpy(output.ptr(),d_output,output.rows*output.cols,hipMemcpyDeviceToHost);
	checkCudaErrors(cudaStatus);	

	cudaStatus = hipFree(d_input);
	checkCudaErrors(cudaStatus);	
	cudaStatus = hipFree(d_output);
	checkCudaErrors(cudaStatus);	
}


